#include "hip/hip_runtime.h"
/**
 * Implementation of Adaptive Stream-based Entropy Coding (ASE-Coding) for CUDA GPUs
 * 
 * このプログラムは, ASE-Coding のリファレンス実装である. GPU のメニーコアを利用して, 並列圧縮を行う.
 * また, NVIDIA CUDA に対応したハードウェア上でのみ動作する.
 * 
 * Author: Saneyuki Tadokoro (201311374)
 */

#include <iostream>
#include <tuple>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include "ase.cuh"

__device__ static const char CMARK_TRUE = 1;
__device__ static const char CMARK_FALSE = 0;

namespace ase {

// デバイスのリセットを行う.
int resetDevice() {
  const hipError_t error = hipDeviceReset();
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 
    fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
    return -1;
  }
  return 0;
}

// スレッド数が1より多い場合は, 入力ストリームを十分に分割できないので, 不足しているスレッド数の計算を行う.
int correct_threads(int threads, int total_size, int chunk_size) {
  int lack_threads = 0;

  if (threads > 1) {
    int remaining = total_size - (total_size / threads) * threads;
    if (remaining <= chunk_size) {
      lack_threads = 1;
    } else {
      lack_threads = (remaining / chunk_size) + 1;
    }
  }

  return threads + lack_threads;
}

CompDescriptor** malloc_comp_descriptors(const Partition *partition, int nread) {
  CompDescriptor **descs = (CompDescriptor**)malloc(partition->grids * sizeof(CompDescriptor));
  for (int i = 0; i < partition->grids; i++) {
    descs[i]->entry_size = partition->allocations[i].entry_size;
    descs[i]->global_counter = partition->allocations[i].global_counter;
    descs[i]->chunk_size = nread / partition->allocations[i].threads;
    descs[i]->total_size = nread / partition->grids;
    descs[i]->threads = correct_threads(partition->allocations[i].threads, descs[i]->total_size, descs[i]->chunk_size);
  }
  return descs;
}

DecompDescriptor** malloc_decomp_descriptors(const Partition *partition, long *counts) {
  DecompDescriptor **descs = (DecompDescriptor**)malloc(partition->grids * sizeof(DecompDescriptor));
  for (int i = 0; i< partition->grids; i++) {
    descs[i]->entry_size = partition->allocations[i].entry_size;
    descs[i]->global_counter = partition->allocations[i].global_counter;
    descs[i]->threads = partition->allocations[i].threads;
    descs[i]->counts = counts;
  }
  return descs;
}

__host__ __device__
Data* malloc_ase_data() {
  Data *new_data;

  if ((new_data = (Data*)malloc(sizeof(Data))) == NULL)
    return NULL;
  new_data->data = 0;
  return new_data;
}

__host__ __device__
Data* malloc_next_ase_data(Buffer* buf) {
  Data *new_data;

  if ((new_data = malloc_ase_data()) == NULL)
    return NULL;
  buf->current->next = new_data;
  buf->current = new_data;
  buf->t_offset = 0;
  return new_data;
}

__host__ __device__
Buffer* malloc_ase_buffer() {
  Buffer *new_buf;
  Data *new_data;

  if ((new_buf = (Buffer*)malloc(sizeof(Buffer))) == NULL)
    return NULL;
  if ((new_data = malloc_ase_data()) == NULL)
    return NULL;
  new_buf->head = new_data;
  new_buf->current = new_data;
  new_buf->max_width = D_MAX_WIDTH;
  new_buf->t_offset = 0;
  new_buf->h_offset = 0;
  return new_buf;
}

__host__ __device__
Buffer** malloc_ase_buffer(int nums) {
  Buffer **new_bufs = (Buffer**)malloc(nums * sizeof(Buffer));
  Data *new_data;
  int i;

  for (i = 0; i < nums; i++) {
    if ((new_data = malloc_ase_data()) == NULL)
      return NULL;
    new_bufs[i]->head = new_data;
    new_bufs[i]->current = new_data;
    new_bufs[i]->max_width = D_MAX_WIDTH;
    new_bufs[i]->t_offset = 0;
    new_bufs[i]->h_offset = 0;
  }
  return new_bufs;
}

__host__ __device__
Context* malloc_ase_context(int global_counter) {
  Context *context;

  if ((context = (Context *)malloc(sizeof(Context))) == NULL)
    return NULL;
  context->occupied = 0;
  context->global_counter = global_counter;
  context->culling_num = global_counter;
  context->max_entropy = 1;
  return context;
}

__host__ __device__
int free_head_ase_buffer(Buffer *buf) {
  Data *new_head;

  if (buf->head == NULL) {
    return -1;
  }

  if (buf->head->next != NULL) {
    new_head = buf->head->next;
    free(buf->head);
    buf->head = new_head;
  } else {
    free(buf->head);
    buf->current = NULL;
    buf->head = NULL;
  }
  buf->h_offset = 0;
  return 0;
}

// 8ビット長のデータバッファに任意サイズの入力データを書き込む関数.
// データバッファには, 上位ビットから入力データを詰める形式で書き込む.
__host__ __device__
int write_data_to_buf(Buffer* buf, const char* data, const unsigned int width) {
  int shift = buf->max_width - buf->t_offset - width;

  // シフト量が 0 より少ない
  if (shift < 0) {
    // 入力データに対する残量シフト量, すなわち -1 * shift に等しい.
    // 注意! unsigned char で論理右シフトを行う.
    buf->current->data |= (unsigned char)*data >> (-1 * shift);

    // 新規バッファ確保
    if (malloc_next_ase_data(buf) == NULL)
      return -1;

    // データの最大の長さにマイナス値の shift を加えると, 残りの書き込み量が計算できる (反転).
    shift += buf->max_width;
  }
  buf->current->data |= *data << shift;
  buf->t_offset = buf->max_width - shift;

  return 0;
}

__host__ __device__
int read_data_from_buf(Buffer* buf, char* data, const unsigned int width) {
  int shift = buf->max_width - buf->h_offset - width;
  int remaining = 0;
  *data = 0;

  if (shift < 0) {
    *data = (buf->head->data & (
      (unsigned char)0xFF >> buf->h_offset
    )) << (-1 * shift);

    // 不要バッファ解放
    if (free_head_ase_buffer(buf) == -1) {
      return -1;
    }
    
    remaining = width + shift;
    shift += buf->max_width;
  }
  *data |= (unsigned char)(
    buf->head->data & (
      (unsigned char)(0xFF << buf->max_width - (width - remaining)) >> buf->h_offset
    )) >> shift;

  buf->h_offset = buf->max_width - shift;

  if (buf->h_offset == buf->max_width) {
    if (free_head_ase_buffer(buf) == -1)
      return -1;
  } 

  return 0;
}

// エントロピーカリングを行う.
__host__ __device__
void entropy_culling(Context *context) {
  if (context->global_counter > 0) {
    context->global_counter--;
  } else {
    if (context->occupied > 0) {
      context->occupied--;
    }
    context->global_counter = context->culling_num;
  }
}

// ヒットしたシンボルがある場合に呼び出される.
// ルックアップテーブル内にヒットしたシンボルは, 最下位エントリにピボットし、
// それ以外のシンボルは上位エントリにピボットする.
__host__ __device__
void arrange_table(Context *context,
                   char *entries,
                   const int hit_index,
                   const char symbol) {
  int i;

  // 占有エントリ数が1でかつ最下位エントリのシンボルがヒットし続ける場合、
  // エントロピーカリングが実行されて占有エントリ数が 0 にデクリメントされないようにする.
  // つまり, この場合は常に最短ビットが生成されることになる.
  if (context->occupied > 1 && hit_index > 0) {
    for (i = hit_index - 1; i >= 0; i--) {
      entries[i + 1] = entries[i];
    }
    entries[0] = symbol;
    entropy_culling(context);
  }
}

// ミスヒットしたシンボルがある場合に呼び出される.
// 最下位エントリにシンボルを追加し, それ以外のシンボルは上位エントリにピボットする.
// ルックアップテーブルのエントリ数がいっぱいになっている場合は, シンボルを追加しない.
__host__ __device__
void register_to_table(Context *context,
                       char *entries,
                       const char symbol) {
  int i;

  for (i = context->occupied - 1; i >= 0; i--) {
    if (i + 1 < E_LENGTH) {
      entries[i + 1] = entries[i];
    }
  }
  entries[0] = symbol;
  if (context->occupied + 1 < E_LENGTH) {
    context->occupied++;
  }
}

// ルックアップテーブルに登録されているシンボルがヒットするかどうかを確認し,
// ヒットすれば該当するエントリインデックスが, ミスヒットすれば-1を返す.
// また, いずれの場合であってもルックアップテーブル操作を試みる.
__host__ __device__
int push(Context *context,
         char *entries,
         const char symbol) {
  int i;

  for (i = 0; i < context->occupied; i++) {
    if (entries[i] == symbol) {
      arrange_table(context, entries, i, symbol);
      return i;
    }
  }
  register_to_table(context, entries, symbol);
  return -1;
}

// 現在の占有エントリ数を用いてエントロピー計算を行う.
__host__ __device__
int entropy_calc(Context *context) {
  int m = (int) ceilf(log2f(context->occupied));
  if (context->max_entropy < m)
    context->max_entropy = m;
  return context->max_entropy;
}

// ASE 圧縮を行うカーネル関数. 入力ストリームを N 分割したストリームをそれぞれのスレッドが ASE 圧縮
// を行う. スレッドが処理すべきデータサイズは ase_settings の chunk_size に定められている.
__global__
void kernel_compress(const char *d_input_data,
                     Buffer **d_out_bufs,
                     long *d_counts,
                     const CompDescriptor **descs) {
  int i, m, hit_index;
  char hit_index_m, symbol;
  const int chunk_size = descs[0]->chunk_size;

  // スレッド番号の計算
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // ルックアップテーブルのエントリ初期化
  char entries[E_LENGTH] = {0};

  Context *context = malloc_ase_context(descs[0]->global_counter);

  for (i = 0; i < chunk_size; i++) {
    // オーバーフローチェック. スレッドが処理すべきデータ範囲を超える場合には処理を終了する.
    if (idx * chunk_size + i > descs[0]->total_size)
      break;

    // シンボルがヒットするかどうかを確認し, ルックアップテーブルを操作する.
    symbol = d_input_data[idx * chunk_size + i];
    hit_index = push(context, entries, symbol);

    // ヒットしなかった場合は, cmark ビットを 0 とし, 圧縮せずにバッファに追加する (シリアライズ).
    if (hit_index == -1) {
      d_counts[idx] = d_counts[idx] + 1 + SYM_SIZE;

      write_data_to_buf(d_out_bufs[idx], &CMARK_FALSE, 1);
      write_data_to_buf(d_out_bufs[idx], &symbol, SYM_SIZE);

    // ヒットした場合は, cmark ビットを 1 とし, 圧縮してシリアライズする.
    } else {
      m = entropy_calc(context);
      hit_index_m = hit_index & ((1 << m) - 1);

      d_counts[idx] = d_counts[idx] + 1 + m;

      write_data_to_buf(d_out_bufs[idx], &CMARK_TRUE, 1);
      write_data_to_buf(d_out_bufs[idx], &hit_index_m, m);
    }
  }
}

// ASE 解凍を行うカーネル関数. 入力ストリームを N 分割したストリームをそれぞれのスレッドが ASE 解凍
// を行う. スレッドが処理すべきデータサイズは ase_settings の chunk_size に定められている.
__global__
void kernel_decompress(Buffer **d_input_bufs,
                       char *d_output_data,
                       long *d_counts,
                       const DecompDescriptor **descs) {
  // int m;
  // int counter = 0;
  // int remaining = settings->bit_size;
  // char index_m, cmark, symbol;
  // char entries[E_LENGTH] = {0};

  // Context *context = malloc_ase_context(settings);

  // read_data_from_buf(input_buf, &cmark, 1);

  // while (true) {
  //   if (cmark == CMARK_TRUE) {
  //     m = entropy_calc(context);

  //     read_data_from_buf(input_buf, &index_m, m);
  //     symbol = entries[index_m];
  //     output_data[counter] = symbol;

  //     arrange_table(context, entries, index_m, symbol);
  //     remaining = remaining - 1 - m;
  //   } else {
  //     read_data_from_buf(input_buf, &symbol, SYM_SIZE);
  //     output_data[counter] = symbol;

  //     register_to_table(context, entries, symbol);
  //     remaining = remaining - 1 - SYM_SIZE;
  //   }

  //   counter++;
  //   *counts = *counts + SYM_SIZE;

  //   if (remaining <= 0)
  //     break;

  //   read_data_from_buf(input_buf, &cmark, 1);
  // }

  // free(context);
}

__host__
void host_compress(const char *input_data,
                   Buffer *out_buf,
                   long *counts,
                   const CompDescriptor *desc) {
  int m, hit_index;
  char hit_index_m, symbol;
  char entries[E_LENGTH] = {0};

  Context *context = malloc_ase_context(desc->global_counter);

  for (int i = 0; i < desc->chunk_size; i++) {
    symbol = input_data[i];
    hit_index = push(context, entries, symbol);

    if (hit_index == -1) {
      *counts = *counts + 1 + SYM_SIZE;

      // CMark (0) ビットの追加
      write_data_to_buf(out_buf, &CMARK_FALSE, 1);
      write_data_to_buf(out_buf, &symbol, SYM_SIZE);
    } else {
      m = entropy_calc(context);

      hit_index_m = hit_index & ((1 << m) - 1);
      *counts = *counts + 1 + m;

      // CMark (1) ビットの追加
      write_data_to_buf(out_buf, &CMARK_TRUE, 1);
      write_data_to_buf(out_buf, &hit_index_m, m);
    }
  }

  free(context);
}

__host__
void host_decompress(Buffer *input_buf,
                     char *output_data,
                     long *counts,
                     const DecompDescriptor *desc) {
  int m;
  int counter = 0;
  int remaining = desc->counts[0];
  char index_m, cmark, symbol;
  char entries[E_LENGTH] = {0};

  Context *context = malloc_ase_context(desc->global_counter);

  read_data_from_buf(input_buf, &cmark, 1);

  while (true) {
    if (cmark == CMARK_TRUE) {
      m = entropy_calc(context);

      read_data_from_buf(input_buf, &index_m, m);
      symbol = entries[index_m];
      output_data[counter] = symbol;

      arrange_table(context, entries, index_m, symbol);
      remaining = remaining - 1 - m;
    } else {
      read_data_from_buf(input_buf, &symbol, SYM_SIZE);
      output_data[counter] = symbol;

      register_to_table(context, entries, symbol);
      remaining = remaining - 1 - SYM_SIZE;
    }

    counter++;
    *counts = *counts + SYM_SIZE;

    if (remaining <= 0)
      break;

    read_data_from_buf(input_buf, &cmark, 1);
  }

  free(context);
}

// ホスト側で ASE 圧縮の準備を行う. 入力ストリームと ASE 設定プロファイルを PCI 転送でデバイスに
// コピーする.
std::tuple<long*, Buffer**> parallel_compress(const char *input_data,
                                              const CompDescriptor **descs,
                                              const Partition *partition) {
  char *d_input_data;
  long *counts, *d_counts;
  const CompDescriptor **d_descs;
  Buffer **d_out_bufs, **out_bufs;

  const int threads = descs[0]->threads;

  // メモリ確保 (ホスト)
  counts = (long*)malloc(threads * sizeof(long));
  out_bufs = malloc_ase_buffer(threads);

  // メモリ確保 (デバイス)
  hipMalloc((void**)&d_input_data, D_SIZE);
  hipMalloc((void**)&d_out_bufs, threads * sizeof(Buffer));
  hipMalloc((void**)&d_descs, sizeof(CompDescriptor));
  hipMalloc((void**)&d_counts, threads * sizeof(long));

  // ホストからデバイスにバス転送
  hipMemcpy(d_input_data, input_data, D_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_descs, descs, sizeof(CompDescriptor), hipMemcpyHostToDevice);
  hipMemcpy(d_out_bufs, out_bufs, sizeof(CompDescriptor), hipMemcpyHostToDevice);

  // カーネル関数呼び出し
  kernel_compress<<<partition->grids, threads>>>(d_input_data, d_out_bufs, d_counts, d_descs);

  // すべてのスレッドが処理を完了するまで待つ
  hipDeviceSynchronize();

  // デバイスリセット
  resetDevice();

  // デバイスからホストにバス転送
  hipMemcpy(out_bufs, d_out_bufs, threads * sizeof(Buffer), hipMemcpyDeviceToHost);
  hipMemcpy(counts, d_counts, threads * sizeof(long), hipMemcpyDeviceToHost);

  // メモリ解放 (デバイス)
  hipFree(d_counts);
  hipFree(d_descs);
  hipFree(d_out_bufs);
  hipFree(d_input_data);

  return {
    counts,
    out_bufs
  };
}

std::tuple<long, char*> parallel_decompress(Buffer **buffer,
                                            const DecompDescriptor **descs,
                                            const Partition *Partition) {
  return {
    0,
    ""
  };
}

std::tuple<long, Buffer*> compress(const char *input_data, const CompDescriptor *desc) {
  long counts = 0;
  Buffer *buffer = malloc_ase_buffer();
  char *output_data = (char*)malloc(D_SIZE * sizeof(char));

  host_compress(input_data, buffer, &counts, desc);

  return {
    counts,
    buffer
  };
}

std::tuple<long, char*> decompress(Buffer *buffer, const DecompDescriptor *desc) {
  long counts = 0;
  char *output_data = (char*)malloc(D_SIZE * sizeof(char));

  host_decompress(buffer, output_data, &counts, desc);

  return {
    counts,
    output_data
  };
}

} // namespace ase
