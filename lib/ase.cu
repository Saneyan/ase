#include "hip/hip_runtime.h"
/**
 * Implementation of Adaptive Stream-based Entropy Coding (ASE-Coding) for CUDA GPUs
 * 
 * このプログラムは, ASE-Coding のリファレンス実装である. GPU のメニーコアを利用して, 並列圧縮を行う.
 * また, NVIDIA CUDA に対応したハードウェア上でのみ動作する.
 * 
 * Author: Saneyuki Tadokoro (201311374)
 */

#include <iostream>
#include <tuple>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include "ase.cuh"

#define C_WORST_RATE 1.1
#define NUM_PARTITIONS_LIMIT 128

static const int D_OUT_SIZE = D_SIZE * C_WORST_RATE;

__device__ static const char CMARK_TRUE = 1;
__device__ static const char CMARK_FALSE = 0;

namespace ase {

// デバイスのリセットを行う.
int resetDevice() {
  const hipError_t error = hipDeviceReset();
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 
    fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
    return -1;
  }
  return 0;
}

// ブロック数が1より多い場合は, 入力ストリームを十分に分割できないので, 不足しているブロック数の計算を行う.
int correct_num_blocks(int num_blocks, int total_size, int chunk_size) {
  int lack_blocks = 0;

  if (num_blocks > 1) {
    int remaining = total_size - (total_size / num_blocks) * num_blocks;
    if (remaining <= chunk_size) {
      lack_blocks= 1;
    } else {
      lack_blocks = (remaining / chunk_size) + 1;
    }
  }

  return num_blocks + lack_blocks;
}

ParallelCompDescriptor* malloc_parallel_comp_descriptors(const Partition *partition, int nread) {
  if (partition->num_allocations > NUM_PARTITIONS_LIMIT) {
    fprintf(stderr, "Cannot apply more than 128 partition allocations.");
    return NULL;
  }

  ParallelCompDescriptor *descs = (ParallelCompDescriptor*)malloc(partition->num_allocations * sizeof(ParallelCompDescriptor));
  int i, num_target_blocks;
  const int chunk_size = nread / partition->num_blocks;
  const int output_size = chunk_size * C_WORST_RATE;
  int remaining_blocks = correct_num_blocks(partition->num_blocks, nread, chunk_size);

  for (i = 0; i < partition->num_allocations; i++) {
    if ((num_target_blocks = partition->allocations[i].num_target_blocks) > 0)
      remaining_blocks -= num_target_blocks;
  }
  for (i = 0; i < partition->num_allocations; i++) {
    num_target_blocks = partition->allocations[i].num_target_blocks;

    descs[i].entry_size = partition->allocations[i].entry_size;
    descs[i].global_counter = partition->allocations[i].global_counter;
    descs[i].chunk_size = chunk_size;
    descs[i].output_size = output_size;
    descs[i].total_size = nread;
    descs[i].num_blocks = num_target_blocks > 0 ? num_target_blocks : remaining_blocks;
  }
  return descs;
}

ParallelDecompDescriptor* malloc_parallel_decomp_descriptors(const Partition *partition, long *counts) {
  ParallelDecompDescriptor *descs = (ParallelDecompDescriptor*)malloc(partition->num_allocations * sizeof(ParallelDecompDescriptor));
  for (int i = 0; i< partition->num_allocations; i++) {
    descs[i].entry_size = partition->allocations[i].entry_size;
    descs[i].global_counter = partition->allocations[i].global_counter;
    descs[i].num_blocks = partition->num_blocks;
    descs[i].counts = counts;
  }
  return descs;
}

__host__
Data* malloc_ase_data() {
  Data *new_data;

  if ((new_data = (Data*)malloc(sizeof(Data))) == NULL)
    return NULL;
  new_data->data = 0;
  return new_data;
}

__host__
Data* malloc_next_ase_data(Buffer* buf) {
  Data *new_data;

  if ((new_data = malloc_ase_data()) == NULL)
    return NULL;
  buf->current->next = new_data;
  buf->current = new_data;
  buf->t_offset = 0;
  return new_data;
}

__host__
Buffer* malloc_ase_buffer() {
  Buffer *new_buf;
  Data *new_data;

  if ((new_buf = (Buffer*)malloc(sizeof(Buffer))) == NULL)
    return NULL;
  if ((new_data = malloc_ase_data()) == NULL)
    return NULL;
  new_buf->head = new_data;
  new_buf->current = new_data;
  new_buf->max_width = D_MAX_WIDTH;
  new_buf->t_offset = 0;
  new_buf->h_offset = 0;
  return new_buf;
}

__host__
int free_head_ase_buffer(Buffer *buf) {
  Data *new_head;

  if (buf->head == NULL) {
    return -1;
  }

  if (buf->head->next != NULL) {
    new_head = buf->head->next;
    free(buf->head);
    buf->head = new_head;
  } else {
    free(buf->head);
    buf->current = NULL;
    buf->head = NULL;
  }
  buf->h_offset = 0;
  return 0;
}

// 8ビット長のデータバッファに任意サイズの入力データを書き込む関数.
// データバッファには, 上位ビットから入力データを詰める形式で書き込む.
__host__
int write_data_to_buf(Buffer* buf, const char* data, const unsigned int width) {
  int shift = buf->max_width - buf->t_offset - width;

  // シフト量が 0 より少ない
  if (shift < 0) {
    // 入力データに対する残量シフト量, すなわち -1 * shift に等しい.
    // 注意! unsigned char で論理右シフトを行う.
    buf->current->data |= (unsigned char)*data >> (-1 * shift);

    // 新規バッファ確保
    if (malloc_next_ase_data(buf) == NULL)
      return -1;

    // データの最大の長さにマイナス値の shift を加えると, 残りの書き込み量が計算できる (反転).
    shift += buf->max_width;
  }
  buf->current->data |= *data << shift;
  buf->t_offset = buf->max_width - shift;

  return 0;
}

__host__
int read_data_from_buf(Buffer* buf, char* data, const unsigned int width) {
  int shift = buf->max_width - buf->h_offset - width;
  int remaining = 0;
  *data = 0;

  if (shift < 0) {
    *data = (buf->head->data & (
      (unsigned char)0xFF >> buf->h_offset
    )) << (-1 * shift);

    // 不要バッファ解放
    if (free_head_ase_buffer(buf) == -1) {
      return -1;
    }
    
    remaining = width + shift;
    shift += buf->max_width;
  }
  *data |= (unsigned char)(
    buf->head->data & (
      (unsigned char)(0xFF << buf->max_width - (width - remaining)) >> buf->h_offset
    )) >> shift;

  buf->h_offset = buf->max_width - shift;

  if (buf->h_offset == buf->max_width) {
    if (free_head_ase_buffer(buf) == -1)
      return -1;
  } 

  return 0;
}

__device__
void next_data(PoolInfo* pi) {
  pi->index++;
  pi->t_offset = 0;
}

__host__
PoolInfo* malloc_pool_info(int nums) {
  PoolInfo* pi = (PoolInfo*)malloc(nums * sizeof(PoolInfo));
  int i;

  for (i = 0; i < nums; i++) {
    pi[i].t_offset = 0;
    pi[i].h_offset = 0;
    pi[i].max_width = D_MAX_WIDTH;
    pi[i].index = 0;
    pi[i].counts = 0;
  }
  return pi;
}

__device__
int write_data_to_pool(PoolInfo* pi, char* pool, const char* data, const unsigned int width) {
  int shift = pi->max_width - pi->t_offset - width;

  if (shift < 0) {
    *pool |= (unsigned char)*data >> (-1 * shift);
    next_data(pi);
    shift += pi->max_width;
  }
  *pool |= *data << shift;
  pi->t_offset = pi->max_width - shift;
  pi->counts += width;

  return 0;
}

// __device__
// int read_data_from_pool(Buffer* buf, char* data, const unsigned int width) {
//   int shift = buf->max_width - buf->h_offset - width;
//   int remaining = 0;
//   *data = 0;

//   if (shift < 0) {
//     *data = (buf->head->data & (
//       (unsigned char)0xFF >> buf->h_offset
//     )) << (-1 * shift);

//     // 不要バッファ解放
//     if (free_head_ase_buffer(buf) == -1) {
//       return -1;
//     }
    
//     remaining = width + shift;
//     shift += buf->max_width;
//   }
//   *data |= (unsigned char)(
//     buf->head->data & (
//       (unsigned char)(0xFF << buf->max_width - (width - remaining)) >> buf->h_offset
//     )) >> shift;

//   buf->h_offset = buf->max_width - shift;

//   if (buf->h_offset == buf->max_width) {
//     if (free_head_ase_buffer(buf) == -1)
//       return -1;
//   } 

//   return 0;
// }

__host__ __device__
Context* malloc_ase_context(int global_counter) {
  Context *context;

  if ((context = (Context *)malloc(sizeof(Context))) == NULL)
    return NULL;
  context->occupied = 0;
  context->global_counter = global_counter;
  context->culling_num = global_counter;
  context->max_entropy = 1;
  return context;
}

// エントロピーカリングを行う.
__host__ __device__
void entropy_culling(Context *context) {
  if (context->global_counter > 0) {
    context->global_counter--;
  } else {
    if (context->occupied > 0) {
      context->occupied--;
    }
    context->global_counter = context->culling_num;
  }
}

// ヒットしたシンボルがある場合に呼び出される.
// ルックアップテーブル内にヒットしたシンボルは, 最下位エントリにピボットし、
// それ以外のシンボルは上位エントリにピボットする.
__host__ __device__
void arrange_table(Context *context,
                   char *entries,
                   const int hit_index,
                   const char symbol) {
  int i;

  // 占有エントリ数が1でかつ最下位エントリのシンボルがヒットし続ける場合、
  // エントロピーカリングが実行されて占有エントリ数が 0 にデクリメントされないようにする.
  // つまり, この場合は常に最短ビットが生成されることになる.
  if (context->occupied > 1 && hit_index > 0) {
    for (i = hit_index - 1; i >= 0; i--) {
      entries[i + 1] = entries[i];
    }
    entries[0] = symbol;
    entropy_culling(context);
  }
}

// ミスヒットしたシンボルがある場合に呼び出される.
// 最下位エントリにシンボルを追加し, それ以外のシンボルは上位エントリにピボットする.
// ルックアップテーブルのエントリ数がいっぱいになっている場合は, シンボルを追加しない.
__host__ __device__
void register_to_table(Context *context,
                       char *entries,
                       const char symbol) {
  int i;

  for (i = context->occupied - 1; i >= 0; i--) {
    if (i + 1 < E_LENGTH) {
      entries[i + 1] = entries[i];
    }
  }
  entries[0] = symbol;
  if (context->occupied + 1 < E_LENGTH) {
    context->occupied++;
  }
}

// ルックアップテーブルに登録されているシンボルがヒットするかどうかを確認し,
// ヒットすれば該当するエントリインデックスが, ミスヒットすれば-1を返す.
// また, いずれの場合であってもルックアップテーブル操作を試みる.
__host__ __device__
int push(Context *context,
         char *entries,
         const char symbol) {
  int i;

  for (i = 0; i < context->occupied; i++) {
    if (entries[i] == symbol) {
      arrange_table(context, entries, i, symbol);
      return i;
    }
  }
  register_to_table(context, entries, symbol);
  return -1;
}

// 現在の占有エントリ数を用いてエントロピー計算を行う.
__host__ __device__
int entropy_calc(Context *context) {
  int m = (int) ceilf(log2f(context->occupied));
  if (context->max_entropy < m)
    context->max_entropy = m;
  return context->max_entropy;
}

// ASE 圧縮を行うカーネル関数. 入力ストリームを N 分割したストリームをそれぞれのスレッドが ASE 圧縮
// を行う. スレッドが処理すべきデータサイズは ase_settings の chunk_size に定められている.
__global__
void kernel_compress(const char *d_input_data,
                     char *d_output_data,
                     PoolInfo *d_pi,
                     const ParallelCompDescriptor *desc,
                     int *target_block_ids) {
  int i, m, hit_index;
  char hit_index_m, symbol;

  const int tid = target_block_ids[blockIdx.x];

  // ルックアップテーブルのエントリ初期化
  char entries[E_LENGTH] = {0};
  Context *context = malloc_ase_context(desc->global_counter);

  for (i = 0; i < desc->chunk_size; i++) {
    // オーバーフローチェック. スレッドが処理すべきデータ範囲を超える場合には処理を終了する.
    if (tid * desc->chunk_size + i > desc->total_size)
      break;

    // シンボルがヒットするかどうかを確認し, ルックアップテーブルを操作する.
    symbol = d_input_data[tid * desc->chunk_size + i];
    hit_index = push(context, entries, symbol);

    // ヒットしなかった場合は, cmark ビットを 0 とし, 圧縮せずにバッファに追加する (シリアライズ).
    if (hit_index == -1) {
      write_data_to_pool(d_pi, &d_output_data[tid * desc->output_size], &CMARK_FALSE, 1);
      write_data_to_pool(d_pi, &d_output_data[tid * desc->output_size], &symbol, SYM_SIZE);

    // ヒットした場合は, cmark ビットを 1 とし, 圧縮してシリアライズする.
    } else {
      m = entropy_calc(context);
      hit_index_m = hit_index & ((1 << m) - 1);

      write_data_to_pool(d_pi, &d_output_data[tid * desc->output_size], &CMARK_TRUE, 1);
      write_data_to_pool(d_pi, &d_output_data[tid * desc->output_size], &hit_index_m, m);
    }
  }
}

// ASE 解凍を行うカーネル関数. 入力ストリームを N 分割したストリームをそれぞれのスレッドが ASE 解凍
// を行う. スレッドが処理すべきデータサイズは ase_settings の chunk_size に定められている.
__global__
void kernel_decompress(Buffer *d_input_bufs,
                       char *d_output_data,
                       long *d_counts,
                       const DecompDescriptor *descs) {
  // int m;
  // int counter = 0;
  // int remaining = settings->bit_size;
  // char index_m, cmark, symbol;
  // char entries[E_LENGTH] = {0};

  // Context *context = malloc_ase_context(settings);

  // read_data_from_buf(input_buf, &cmark, 1);

  // while (true) {
  //   if (cmark == CMARK_TRUE) {
  //     m = entropy_calc(context);

  //     read_data_from_buf(input_buf, &index_m, m);
  //     symbol = entries[index_m];
  //     output_data[counter] = symbol;

  //     arrange_table(context, entries, index_m, symbol);
  //     remaining = remaining - 1 - m;
  //   } else {
  //     read_data_from_buf(input_buf, &symbol, SYM_SIZE);
  //     output_data[counter] = symbol;

  //     register_to_table(context, entries, symbol);
  //     remaining = remaining - 1 - SYM_SIZE;
  //   }

  //   counter++;
  //   *counts = *counts + SYM_SIZE;

  //   if (remaining <= 0)
  //     break;

  //   read_data_from_buf(input_buf, &cmark, 1);
  // }

  // free(context);
}

__host__
void host_compress(const char *input_data,
                   Buffer *out_buf,
                   long *counts,
                   const CompDescriptor *desc) {
  int m, hit_index;
  char hit_index_m, symbol;
  char entries[E_LENGTH] = {0};

  Context *context = malloc_ase_context(desc->global_counter);

  for (int i = 0; i < desc->chunk_size; i++) {
    symbol = input_data[i];
    hit_index = push(context, entries, symbol);

    if (hit_index == -1) {
      *counts = *counts + 1 + SYM_SIZE;

      // CMark (0) ビットの追加
      write_data_to_buf(out_buf, &CMARK_FALSE, 1);
      write_data_to_buf(out_buf, &symbol, SYM_SIZE);
    } else {
      m = entropy_calc(context);

      hit_index_m = hit_index & ((1 << m) - 1);
      *counts = *counts + 1 + m;

      // CMark (1) ビットの追加
      write_data_to_buf(out_buf, &CMARK_TRUE, 1);
      write_data_to_buf(out_buf, &hit_index_m, m);
    }
  }

  free(context);
}

__host__
void host_decompress(Buffer *input_buf,
                     char *output_data,
                     long *counts,
                     const DecompDescriptor *desc) {
  int m;
  int counter = 0;
  int remaining = desc->counts;
  char index_m, cmark, symbol;
  char entries[E_LENGTH] = {0};

  Context *context = malloc_ase_context(desc->global_counter);

  read_data_from_buf(input_buf, &cmark, 1);

  while (true) {
    if (cmark == CMARK_TRUE) {
      m = entropy_calc(context);

      read_data_from_buf(input_buf, &index_m, m);
      symbol = entries[index_m];
      output_data[counter] = symbol;

      arrange_table(context, entries, index_m, symbol);
      remaining = remaining - 1 - m;
    } else {
      read_data_from_buf(input_buf, &symbol, SYM_SIZE);
      output_data[counter] = symbol;

      register_to_table(context, entries, symbol);
      remaining = remaining - 1 - SYM_SIZE;
    }

    counter++;
    *counts = *counts + SYM_SIZE;

    if (remaining <= 0)
      break;

    read_data_from_buf(input_buf, &cmark, 1);
  }

  free(context);
}

// ホスト側で ASE 圧縮の準備を行う. 入力ストリームと ASE 設定プロファイルを PCI 転送でデバイスに
// コピーする.
std::tuple<PoolInfo*, char*> parallel_compress(const char *input_data,
                                             const ParallelCompDescriptor *descs,
                                             const Partition *partition) {
  int i, j, k, l;
  int *used_block_ids, *d_target_block_ids[NUM_PARTITIONS_LIMIT], *target_block_ids[NUM_PARTITIONS_LIMIT];
  char *d_input_data, *d_output_data, *output_data;
  ParallelCompDescriptor *d_descs[NUM_PARTITIONS_LIMIT];
  PoolInfo *d_pi, *out_pi, *pi;

  // メモリ確保 (ホスト)
  pi = malloc_pool_info(partition->num_blocks);
  output_data = (char*)malloc(D_OUT_SIZE);
  out_pi = (PoolInfo*)malloc(partition->num_blocks * sizeof(PoolInfo));
  used_block_ids = (int*)malloc(partition->num_blocks * sizeof(int));
  int u = 0;

  for (i = 0; i < partition->num_allocations; i++) {
    target_block_ids[i] = (int*)malloc(descs[i].num_blocks * sizeof(int));
  }
  for (i = 0; i < partition->num_allocations; i++) {
    for (j = 0; j < partition->num_blocks; j++) {
      if (partition->allocations[i].num_target_blocks > 0) {
        for (k = 0; k < partition->allocations[i].num_target_blocks; k++) {
          if (partition->allocations[i].target_block_ids[k] == j) {
            target_block_ids[i][k] = j;
            used_block_ids[u] = j;
            u++;
          }
        }
      }
    }
  }

  bool found = false;

  for (i = 0; i < partition->num_allocations; i++) {
    for (j = 0; j < partition->num_blocks; j++) {
      if (partition->allocations[i].num_target_blocks == 0) {
        for (k = 0; k < descs[i].num_blocks; k++) {
          for (l = 0; l < u; l++) {
            if (used_block_ids[l] == j) {
              found = true;
              break;
            }
          }
          if (!found) {
            target_block_ids[i][k] = j;
            found = false;
          }
        }
      }
    }
  }

  // メモリ確保 (デバイス)
  hipMalloc((void**)&d_input_data, D_SIZE);
  hipMalloc((void**)&d_output_data, D_OUT_SIZE);
  hipMalloc((void**)&d_pi, partition->num_blocks * sizeof(PoolInfo));

  // ホストからデバイスにバス転送
  hipMemcpy(d_input_data, input_data, D_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_pi, pi, partition->num_blocks * sizeof(PoolInfo), hipMemcpyHostToDevice);

  // 並行カーネル関数呼び出し (non-blocking)
  for (i = 0; i < partition->num_allocations; i++) {
    hipMalloc((void**)&d_descs[i], sizeof(ParallelCompDescriptor));
    hipMalloc((void**)&d_target_block_ids[i], descs[i].num_blocks * sizeof(int));
    hipMemcpy(d_descs[i], &descs[i], sizeof(ParallelCompDescriptor), hipMemcpyHostToDevice);
    hipMemcpy(d_target_block_ids[i], target_block_ids[i], descs[i].num_blocks * sizeof(int), hipMemcpyHostToDevice);

    kernel_compress<<<descs[i].num_blocks, 1>>>(d_input_data, d_output_data, d_pi, d_descs[i], d_target_block_ids[i]);
  }

  // すべてのスレッドが処理を完了するまで待つ
  hipDeviceSynchronize();

  // デバイスからホストにバス転送
  hipMemcpy(output_data, d_output_data, D_OUT_SIZE, hipMemcpyDeviceToHost);
  hipMemcpy(out_pi, d_pi, partition->num_blocks * sizeof(PoolInfo), hipMemcpyDeviceToHost);

  // アロケーションごとのメモリ解放
  for (i = 0; i < partition->num_allocations; i++) {
    free(target_block_ids[i]);
    hipFree(d_descs[i]);
    hipFree(d_target_block_ids[i]);
  }

  // メモリ解放 (デバイス)
  hipFree(d_output_data);
  hipFree(d_pi);
  hipFree(d_input_data);

std::cout << out_pi->counts / 8 <<std::endl;

  // デバイスリセット
  resetDevice();

  return {
    d_pi,
    output_data
  };
}

std::tuple<long, char*> parallel_decompress(Buffer *buffer,
                                            const ParallelDecompDescriptor *descs,
                                            const Partition *Partition) {
  char a[] = "";
  return {
    0,
    a
  };
}

std::tuple<long, Buffer*> compress(const char *input_data, const CompDescriptor *desc) {
  long counts = 0;
  Buffer *buffer = malloc_ase_buffer();
  char *output_data = (char*)malloc(D_SIZE * sizeof(char));

  host_compress(input_data, buffer, &counts, desc);

  return {
    counts,
    buffer
  };
}

std::tuple<long, char*> decompress(Buffer *buffer, const DecompDescriptor *desc) {
  long counts = 0;
  char *output_data = (char*)malloc(D_SIZE * sizeof(char));

  host_decompress(buffer, output_data, &counts, desc);

  return {
    counts,
    output_data
  };
}

} // namespace ase
