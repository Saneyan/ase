#include "hip/hip_runtime.h"
/**
 * Implementation of Adaptive Stream-based Entropy Coding (ASE-Coding) for CPUs.
 * 
 * このプログラムは, ASE-Coding のリファレンス実装である.
 * 詳細は GPU 版のソースコードをご覧いただきたい.
 * 
 * Date: 2022/2/2
 * Author: Saneyuki Tadokoro (201311374)
 * Version: v0.0.1
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

typedef char *ase_sym;

#define D_SIZE 50 * 1024 * 1000
#define E_LENGTH 8
#define GLOBAL_C 4
#define SYM_SIZE 8
#define D_MAX_WIDTH 8

// ref: https://stackoverflow.com/questions/111928/is-there-a-printf-converter-to-print-in-binary-format
// printf("Leading text "BYTE_TO_BINARY_PATTERN, BYTE_TO_BINARY(*data));
#define BYTE_TO_BINARY_PATTERN "%c%c%c%c%c%c%c%c"
#define BYTE_TO_BINARY(byte)  \
  (byte & 0x80 ? '1' : '0'), \
  (byte & 0x40 ? '1' : '0'), \
  (byte & 0x20 ? '1' : '0'), \
  (byte & 0x10 ? '1' : '0'), \
  (byte & 0x08 ? '1' : '0'), \
  (byte & 0x04 ? '1' : '0'), \
  (byte & 0x02 ? '1' : '0'), \
  (byte & 0x01 ? '1' : '0')

static const char CMARK_TRUE = 1;
static const char CMARK_FALSE = 0;

struct ase_settings {
  int entry_size;
  int global_counter;
  int chunk_size;
  int total_size;
  int bit_size;
};

struct ase_context {
  int occupied;
  int global_counter;
  int culling_num;
};

// データバッファノード
struct ase_data {
  char data;
  struct ase_data* next;
};

// データバッファ (Linked list)
struct ase_buffer {
  unsigned int h_offset;  // 1 ノードに対して書き込んだビット数
  unsigned int t_offset;
  unsigned int max_width; // データの最大の長さ
  struct ase_data* head;
  struct ase_data* current;
};

ase_data* malloc_ase_data() {
  ase_data *new_data;

  if ((new_data = (ase_data*)malloc(sizeof(ase_data))) == NULL) {
    fprintf(stderr, "Cannot allocate memory for ase data.\n");
    return NULL;
  }
  new_data->data = 0;
  return new_data;
}

ase_data* malloc_next_ase_data(ase_buffer* buf) {
  ase_data *new_data;

  if ((new_data = malloc_ase_data()) == NULL)
    return NULL;
  buf->current->next = new_data;
  buf->current = new_data;
  buf->t_offset = 0;
  return new_data;
}

ase_buffer* malloc_ase_buffer() {
  ase_buffer *new_buf;
  ase_data *new_data;

  if ((new_buf = (ase_buffer*)malloc(sizeof(ase_buffer))) == NULL) {
    fprintf(stderr, "Cannot allocate memory for ase buffer.\n");
    return NULL;
  }
  if ((new_data = malloc_ase_data()) == NULL)
    return NULL;
  new_buf->head = new_data;
  new_buf->current = new_data;
  new_buf->max_width = D_MAX_WIDTH;
  new_buf->t_offset = 0;
  new_buf->h_offset = 0;
  return new_buf;
}

ase_context* malloc_ase_context(const ase_settings* settings) {
  ase_context *context;

  if ((context = (ase_context *)malloc(sizeof(ase_context))) == NULL) {
    fprintf(stderr, "Cannot allocate memory for ase context.\n");
    return NULL;
  }
  context->occupied = 0;
  context->global_counter = settings->global_counter;
  context->culling_num = settings->global_counter;
  return context;
}

int free_head_ase_buffer(ase_buffer *buf) {
  ase_data *new_head;

  if (buf->head == NULL) {
    return -1;
  }

  if (buf->head->next != NULL) {
    new_head = buf->head->next;
    free(buf->head);
    buf->head = new_head;
  } else {
    free(buf->head);
    buf->current = NULL;
    buf->head = NULL;
  }
  buf->h_offset = 0;
  return 0;
}

// 8ビット長のデータバッファに任意サイズの入力データを書き込む関数.
// データバッファには, 上位ビットから入力データを詰める形式で書き込む.
int data_to_buf(ase_buffer* buf, const char* data, const unsigned int width) {
  int shift = buf->max_width - buf->t_offset - width;

  if (shift < 0) {
    // 入力データに対する残量シフト量, すなわち -1 * shift に等しい.
    buf->current->data |= (unsigned char)*data >> (-1 * shift);

    // 新規バッファ確保
    if (malloc_next_ase_data(buf) == NULL)
      return -1;

    // データの最大の長さにマイナス値の shift を加えると, 残りの書き込み量が計算できる (反転).
    shift += buf->max_width;
  }
  buf->current->data |= *data << shift;
  buf->t_offset = buf->max_width - shift;

  return 0;
}

int data_from_buf(ase_buffer* buf, char* data, const unsigned int width) {
  int shift = buf->max_width - buf->h_offset - width;
  int remaining = 0;
  *data = 0;

  if (shift < 0) {
    *data = (buf->head->data & ((unsigned char)0xFF >> buf->h_offset)) << (-1 * shift);

    // 不要バッファ解放
    if (free_head_ase_buffer(buf) == -1) {
      return -1;
    }
    
    remaining = width + shift;
    shift += buf->max_width;
  }
  *data |= (unsigned char)(buf->head->data & ((unsigned char)(0xFF << buf->max_width - (width - remaining)) >> buf->h_offset)) >> shift;
  buf->h_offset = buf->max_width - shift;

  if (buf->h_offset == buf->max_width) {
    if (free_head_ase_buffer(buf) == -1)
      return -1;
  } 

  return 0;
}

void entropy_culling(ase_context *context) {
  if (context->global_counter > 0) {
    context->global_counter--;
  } else {
    if (context->occupied > 0)
      context->occupied--;
    context->global_counter = context->culling_num;
  }
}

void arrange_table(ase_context *context,
                   char *entries,
                   const int hit_index,
                   const char symbol) {
  int i;

  if (context->occupied > 1 && hit_index > 0) {
    for (i = hit_index - 1; i >= 0; i--)
      entries[i + 1] = entries[i];

    entries[0] = symbol;
    entropy_culling(context);
  }
}

void register_to_table(ase_context *context, char *entries, const char symbol) {
  int i;

  for (i = context->occupied - 1; i >= 0; i--) {
    if (i + 1 < E_LENGTH)
      entries[i + 1] = entries[i];
  }
  entries[0] = symbol;
  if (context->occupied + 1 < E_LENGTH)
    context->occupied++;
}

int push(ase_context *context, char *entries, const char symbol) {
  int i;

  for (i = 0; i < context->occupied; i++) {
    if (entries[i] == symbol) {
      arrange_table(context, entries, i, symbol);
      return i;
    }
  }
  register_to_table(context, entries, symbol);
  return -1;
}

int entropy_calc(ase_context *context) {
  int m = (int) ceilf(log2f(context->occupied));
  return m == 0 ? 1 : m;
}

void ase_compress(const char *input_data,
                  ase_buffer *out_buf,
                  long *counts,
                  const ase_settings *settings) {
  int m;
  int hit_index;
  char hit_index_m;
  int max_m = 0;
  char symbol;
  char entries[E_LENGTH] = {0};
  ase_context *context = malloc_ase_context(settings);

  for (int i = 0; i < settings->chunk_size; i++) {
    symbol = input_data[i];
    hit_index = push(context, entries, symbol);

    if (hit_index == -1) {
      *counts = *counts + 1 + SYM_SIZE;

      // CMark (0) ビットの追加
      data_to_buf(out_buf, &CMARK_FALSE, 1);
      data_to_buf(out_buf, &symbol, SYM_SIZE);
    } else {
      m = entropy_calc(context);
      if (max_m < m)
        max_m = m;

      hit_index_m = hit_index & ((1 << max_m) - 1);
      *counts = *counts + 1 + max_m;

      // CMark (1) ビットの追加
      data_to_buf(out_buf, &CMARK_TRUE, 1);
      data_to_buf(out_buf, &hit_index_m, max_m);
    }
  }

  free(context);
}

void ase_decompress(ase_buffer *input_buf,
                    char *output_data,
                    long *counts,
                    const ase_settings *settings) {
  int m;
  int counter = 0;
  int remaining = settings->bit_size;
  int index;
  int max_m = 0;
  char index_m, cmark, symbol;
  char entries[E_LENGTH] = {0};
  ase_context *context = malloc_ase_context(settings);

  data_from_buf(input_buf, &cmark, 1);

  while (true) {
    if (cmark == CMARK_TRUE) {
      m = entropy_calc(context);
      if (max_m < m)
        max_m = m;

      data_from_buf(input_buf, &index_m, max_m);
      symbol = entries[index_m];
      output_data[counter] = symbol;

      arrange_table(context, entries, index_m, symbol);
      remaining = remaining - 1 - max_m;
    } else {
      data_from_buf(input_buf, &symbol, SYM_SIZE);
      output_data[counter] = symbol;

      register_to_table(context, entries, symbol);
      remaining = remaining - 1 - SYM_SIZE;
    }

    counter++;
    *counts = *counts + SYM_SIZE;

    if (remaining <= 0) {
      printf("counts: %d\n", *counts);
      printf("remaining: %d\n", remaining);
      break;
    }

    data_from_buf(input_buf, &cmark, 1);
  }

  free(context);
}

long start_compress(const char *input_data, const ase_settings *settings) {
  long counts = 0;
  long out_counts = 0;
  ase_buffer *buffer = malloc_ase_buffer();
  char *output_data = (char*)malloc(D_SIZE * sizeof(char));

  ase_compress(input_data, buffer, &counts, settings);

  ase_settings *o_settings = (ase_settings *)malloc(sizeof(ase_settings));
  o_settings->entry_size = E_LENGTH;
  o_settings->bit_size = counts;
  o_settings->global_counter = GLOBAL_C;


  ase_decompress(buffer, output_data, &out_counts, o_settings);
  FILE *file = fopen("result.iso", "w+");
  fwrite(output_data, 1, settings->total_size, file);
  fclose(file);

  printf("After compressed: %d\n", counts / 8);
  printf("After decompressed: %d\n", out_counts / 8);

  free(buffer);

  return counts / 8 + (counts % 8 > 0 ? 1 : 0);
}

long timer() {
  struct timespec ts;
	struct tm tm;

  clock_gettime(CLOCK_REALTIME, &ts);
  localtime_r(&ts.tv_sec, &tm);
  return (ts.tv_sec * 1000) + (ts.tv_nsec / 1000000);
}

int main(int argc, char **argv) {
  char *input_data;
  ase_settings *settings;
  size_t nread;
  long b_time, a_time, total_size;
  const char filename[] = "./test.iso";

  input_data = (char*)malloc(D_SIZE * sizeof(char));

  FILE *file = fopen(filename, "r");
  if (file) {
    while ((nread = fread(input_data, 1, D_SIZE * sizeof(char), file)) > 0) {
      // printf("%zu\n", nread);

      b_time = timer();

      settings = (ase_settings *)malloc(sizeof(ase_settings));
      settings->entry_size = E_LENGTH;
      settings->chunk_size = nread;
      settings->total_size = nread;
      settings->global_counter = GLOBAL_C;

      total_size = start_compress(input_data, settings);

      a_time = timer();

      printf("%s\n", filename);
      printf("Raw: %ld bytes\n", nread);
      printf("Compressed: %ld bytes\n", total_size);
      printf("Compression rate: %f%%\n", ((float)total_size / (float)nread) * 100);
      printf("Timer: %ld msec\n", a_time - b_time);

      free(input_data);
      free(settings);
    }
    fclose(file);
  } else {
    fprintf(stderr, "Cannot open this file.\n");
    return 1;
  }

  return 0;
}
